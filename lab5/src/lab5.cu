
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cmath>
#include <iostream>
#include <sys/time.h>

#define TIME_RESOLUTION 1000000	// time measuring resolution (us)

#define NUM_BLOCKS 128				// this is wrong
#define NUM_THREADS_PER_BLOCK 256	// this is wrong
#define SIZE 2048
#define TILE_SIZE 16

using namespace std;

float m1[SIZE][SIZE], m2[SIZE][SIZE], result[SIZE][SIZE], _result[SIZE][SIZE];
long long unsigned cpu_time;
timeval t;

void startTime (void) {
    gettimeofday(&t, NULL);
    cpu_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

void stopTime (void) {
    gettimeofday(&t, NULL);
    long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

    final_time -= cpu_time;

    cout << final_time << " us have elapsed for the CPU execution" << endl;
}

void fillMatrices (void) {

    for (unsigned i = 0; i < SIZE; ++i) {
        for (unsigned j = 0; j < SIZE; ++j) {
            result[i][j] = 0.0;
            _result[i][j] = 0.0;
            m1[i][j] = ((float) rand()) / ((float) RAND_MAX);
            m2[i][j] = ((float) rand()) / ((float) RAND_MAX);
        }
    }
}

void checkCUDAError (const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
        exit(-1);
    }
}

// You need to optimize AND parallelize this first
void regularMatrixMult (void) {
    for (unsigned i = 0; i < SIZE; ++i) {
        for (unsigned j = 0; j < SIZE; ++j) {

            result[i][j] = 0;
            for (unsigned k = 0; k < SIZE; ++k) {
                result[i][j] += m1[i][k] * m2[k][j];
            }
        }
    }
}

void tiledMatrixMult (void) {
    for (unsigned m = 0; m < SIZE; m += TILE_SIZE) {
        for (unsigned n = 0; n < SIZE; n += TILE_SIZE) {
            //...
        }
    }
}

// Fill the input parameters and kernel qualifier
__global__
void matrixMultKernel (double *dev_m1, double *dev_m2, double *dev_res) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;



}

// Fill with the code required for the GPU stencil (mem allocation, transfers, kernel launch of stencilKernel)
double* matrixMultGPU (void) {
    // you can either:
    // 1 - use 2D matrices, as in CPU
    // 2 - use 1D matrices, but YOU have to convert them here

    return NULL;
}

int main (int argc, char** argv) {

    fillMatrices ();

    // GPU stuff
    matrixMultGPU ();

    // CPU stuff
    startTime();
    regularMatrixMult ();
    stopTime();


    return 0;
}
