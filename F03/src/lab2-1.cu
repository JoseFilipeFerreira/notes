/**************************************************************
 *
 * --== Simple CUDA kernel ==--
 * author: ampereira
 *
 *
 * Fill the rest of the code
 *
 * Insert the functions for time measurement in the correct
 * sections (i.e. do not account for filling the vectors with random data)
 *
 *
 * The stencil array size must be set to the SIZE #define, i.e., float stencil[SIZE];
 **************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <omp.h>

#define TIME_RESOLUTION 1000000	// time measuring resolution (us)

#define NUM_BLOCKS 2560
#define NUM_THREADS_PER_BLOCK 5120
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK
#define NEIGHBOURS 4

using namespace std;

long long unsigned cpu_time;
hipEvent_t start, stop;
timeval t;

void startTime (void) {
    gettimeofday(&t, NULL);
    cpu_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

void stopTime (void) {
    gettimeofday(&t, NULL);
    long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

    final_time -= cpu_time;

    cout << final_time << " us have elapsed for the CPU execution" << endl;
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
}

void stopKernelTime (void) {
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
        exit(-1);
    }
}

// Fill the input parameters and kernel qualifier
__global__
void stencilKernel (double *dev_array, double *dev_output) {
    // the id of the current thread on the GPU - only valid for 1D thread structure
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // REMEMBER: code this as it should only computes a single element of dev_array
    // this kernel will be executed by all CUDA threads that have different ids
    dev_output[id] = 0.0;
    for (int j = -NEIGHBOURS; j <= NEIGHBOURS; j++) {
        // do not do anything if the array is out of bounds
        if (id + j >= 0 &&  id + j < SIZE) {
            dev_output[id] += dev_array[id+j];
        }
    }

}

// Fill with the code required for the GPU stencil (mem allocation, transfers, kernel launch of stencilKernel)
double* stencilGPU (double *array) {
    double *dev_array, *dev_output;
    double *array_output = new double [SIZE];

    // allocate the memory on the device
    hipMalloc((void**) &dev_array, sizeof(double) * SIZE);
    hipMalloc((void**) &dev_output, sizeof(double) * SIZE);

    startKernelTime();
    // copy inputs to the device
    hipMemcpy(dev_array, array, sizeof(double) * SIZE, hipMemcpyHostToDevice);

    // launch the kernel
    stencilKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>> (dev_array, dev_output);

    // copy the output to the host
    hipMemcpy(array_output, dev_output, sizeof(double) * SIZE, hipMemcpyDeviceToHost);
    stopKernelTime();

    // free the device memory
    hipFree(dev_array);
    hipFree(dev_output);

    return array_output;
}

// The CPU stencil - add a value with its 4 neighbours
double* stencilCPU (double *array) {
    double *output_array = new double [SIZE];

    #pragma omp parallel for
    for (int i = 0; i < SIZE; i++) {
        output_array[i] = 0.0;

        for (int j = -NEIGHBOURS; j <= NEIGHBOURS; j++) {
            // do not do anything if the array is out of bounds
            if (i + j >= 0 &&  i + j < SIZE) {
                output_array[i] += array[i+j];
            }
        }
    }

    return output_array;
}

int main (int argc, char** argv) {

    double array [SIZE];

    // initialize array with random values
    for (unsigned i = 0; i < SIZE; i++)
        array[i] = ((double) rand()) / ((double) RAND_MAX);


    stencilGPU(array);

    startTime();
    stencilCPU(array);
    stopTime();


    return 0;
}
